
#include <hip/hip_runtime.h>


#include <stdio.h>
#include <stdlib.h>


#define N 1920;
#define M 1080;

void colorTogrey_BLUR(int *, int *,int,int);

// we have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__ 
void blurKernel(int *Pin_d,  int *Pout_d_B, 
                          int width, int height, int BLUR_SIZE) {

 int Col = threadIdx.x + blockIdx.x * blockDim.x;
 int Row = threadIdx.y + blockIdx.y * blockDim.y;

   unsigned   int pixVal = 0;
   unsigned   int pixels = 0;
 
if (Col < (width) && Row < height) {

     for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) {
     for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) {
        int curRow = Row + blurRow;
        int curCol = Col + blurCol;
          // Verify we have a valid image pixel
        if(curRow > -1 && curRow < height && curCol > -1 && curCol < width) {
           pixVal += Pin_d[curRow * width + curCol];
           pixels++; // Keep track of number of pixels in the avg
            }
          }
        }
      // Write our new pixel value out
  	 Pout_d_B[Row * width + Col] = int(pixVal / pixels);
    }

}


int main()

{
    int n=N;int m=M; 

    int *Pin_h = (int*) malloc( sizeof(int)*n*m);

    int ind=0;
    unsigned int tmp;
    FILE *fp;
    fp=fopen("testImage_Results_RGB.txt","r");
    
    while (!feof(fp)){
        	
        	fscanf(fp,"%d",&tmp);
        	
        	Pin_h[ind]=tmp;
        	
        	ind=ind+1;
        	
            }
            
	fclose(fp);

int *Pout_h_B = (int*) malloc( sizeof(int)*n*m); //for BLUR operation

colorTogrey_BLUR ( Pin_h, Pout_h_B, n, m);

FILE *fp4;
    fp4=fopen("testImageResults_BLUR_RGB.txt","w");
    
    for (int i=0; i < m; i++){
    for (int j=0; j < n; j++){
    fprintf(fp4,"%4d ",Pout_h_B[i*n+j]);}
    fprintf(fp4,"\n");
    }
fclose(fp4);

// free the memory we allocated on the CPU
    free( Pin_h);
    free( Pout_h_B );
        
    return 0;

}



void colorTogrey_BLUR(int *Pin_h, int *Pout_h_B, int n, int m)
{

int size_in = (n *m*sizeof(int)); int size_out = (n*m*sizeof(int));
int *Pin_d; int *Pout_d_B; 
int BLUR_SIZE = 7;

// Transfer Pin_h to device memory 
    hipMalloc((void **) &Pin_d, size_in);
    hipMemcpy(Pin_d, Pin_h, size_in, hipMemcpyHostToDevice);
      
     // Allocate device memory for Pout_d 
    hipMalloc((void **) &Pout_d_B, size_out);

dim3 dimGrid(ceil(n/16), ceil(m/16), 1);
dim3 dimBlock(16,16,1);        

blurKernel<<<dimGrid,dimBlock>>>(Pin_d, Pout_d_B, n, m,BLUR_SIZE);

// Transfer Pout_d from device to host
    
hipMemcpy(Pout_h_B, Pout_d_B, size_out, hipMemcpyDeviceToHost);//copy Blurred Images to Host Memory
     
    
// Free device memory 
     hipFree(Pin_d); hipFree(Pout_d_B);
}



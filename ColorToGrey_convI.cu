
#include <hip/hip_runtime.h>


#include <stdio.h>
#include <stdlib.h>



#define N 1920;
#define M 1080;
#define CHANNELS 3;

void colorTogrey(int *, int *,int,int,int);

// we have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__ 
void colorToGreyscaleConvertion(int *Pin_d,  int *Pout_d,
                          int width, int height) {

 int Col = threadIdx.x + blockIdx.x * blockDim.x;
 int Row = threadIdx.y + blockIdx.y * blockDim.y;
 
 if (Col < (width) && Row < height) {
    // get 1D coordinate for the grayscale image
    int greyOffset = Row*width + Col;
    // one can think of the RGB image having
    // CHANNEL times columns of the gray scale image
    int rgbOffset = greyOffset*3;
    unsigned int r = Pin_d[rgbOffset    ]; // red value for pixel
    unsigned int g = Pin_d[rgbOffset + 1]; // green value for pixel
    unsigned int b = Pin_d[rgbOffset + 2]; // blue value for pixel
    // perform the rescaling and store it
    // We multiply by floating point constants
    Pout_d[greyOffset] = 0.21f*r + 0.72f*g + 0.07f*b;
 }
}


int main()

{
    int n=N;int m=M; int c=CHANNELS;

    int *Pin_h = (int*) malloc( sizeof(int)*n*m*c);

    int ind=0;
    unsigned int tmp;
    FILE *fp;
    fp=fopen("test_image_RGB.txt","r");
    
    while (!feof(fp)){
        	
        	fscanf(fp,"%d",&tmp);
        	
        	Pin_h[ind]=tmp;
        	
        	ind=ind+1;
        	
            }
            
	fclose(fp);

int *Pout_h = (int*) malloc( sizeof(int)*n*m);

colorTogrey ( Pin_h, Pout_h, n, m, c);

FILE *fp3;
    fp3=fopen("testImage_Results_RGB.txt","w");
    
    for (int i=0; i < m; i++){
    for (int j=0; j < n; j++){
    fprintf(fp3,"%4d ",Pout_h[i*n+j]);}
    fprintf(fp3,"\n");
    }
fclose(fp3);

// free the memory we allocated on the CPU
    free( Pin_h);
    free( Pout_h );
        
    return 0;

}



void colorTogrey(int *Pin_h, int *Pout_h, int n, int m, int c)
{

int size_in = (n *m* c*sizeof(int)); int size_out = (n*m*sizeof(int));
int *Pin_d; int *Pout_d; 

// Transfer Pin_h to device memory 
    hipMalloc((void **) &Pin_d, size_in);
    hipMemcpy(Pin_d, Pin_h, size_in, hipMemcpyHostToDevice);
      
     // Allocate device memory for Pout_d 
     hipMalloc((void **) &Pout_d, size_out);

dim3 dimGrid(ceil(n/16), ceil(m/16), 1);
dim3 dimBlock(16,16,1);        

colorToGreyscaleConvertion<<<dimGrid,dimBlock>>>(Pin_d, Pout_d, n, m);

// Transfer Pout_d from device to host
     hipMemcpy(Pout_h, Pout_d, size_out, hipMemcpyDeviceToHost);
     
    
// Free device memory for A_d, B_d, C_d
     hipFree(Pin_d); hipFree(Pout_d); 

}


